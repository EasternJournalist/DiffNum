#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <DiffNum_cuda.h>
#include <cuda_test.h>
#include <vector>

using namespace DiffNum;

__global__ void kernel(ddouble_cuda<2> a, ddouble_cuda<2> b, ddouble_cuda<2>* c) {
    *c = a + Math_cuda<ddouble_cuda<2>>::Sin(b);
    return;
}


ddouble_cuda<2> cuda_test(ddouble_cuda<2> a, ddouble_cuda<2> b) {
    ddouble_cuda<2>* dev_c;
    a.setVar(0), b.setVar(1);
  
    hipMalloc((void**)&dev_c, sizeof(ddouble_cuda<2>));

    kernel<<<1, 1>>>(a, b, dev_c);

    ddouble_cuda<2> c;
    hipMemcpy(&c, dev_c, sizeof(ddouble_cuda<2>), hipMemcpyDeviceToHost);
    hipFree(dev_c);

    return c;
}